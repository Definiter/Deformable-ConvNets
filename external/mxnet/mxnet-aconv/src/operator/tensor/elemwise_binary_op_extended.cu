#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2016 by Contributors
 * \file elemwise_binary_scalar_op.cu
 * \brief GPU Implementation of unary function.
 */
#include "./elemwise_unary_op.h"
#include "./elemwise_binary_op.h"

namespace mxnet {
namespace op {
NNVM_REGISTER_OP(_power)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, mshadow_op::power>);

NNVM_REGISTER_OP(_backward_power)
.set_attr<FCompute>("FCompute<gpu>", BinaryBackwardUseIn<gpu, mshadow_op::power_grad,
                                                              mshadow_op::power_rgrad>);

NNVM_REGISTER_OP(_maximum)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, mshadow_op::maximum>);

NNVM_REGISTER_OP(_backward_maximum)
.set_attr<FCompute>("FCompute<gpu>", BinaryBackwardUseIn<gpu, mshadow_op::ge,
                                                              mshadow_op::lt>);

NNVM_REGISTER_OP(_minimum)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, mshadow_op::minimum>);

NNVM_REGISTER_OP(_backward_minimum)
.set_attr<FCompute>("FCompute<gpu>", BinaryBackwardUseIn<gpu, mshadow_op::le,
                                                              mshadow_op::gt>);

NNVM_REGISTER_OP(_hypot)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, mshadow_op::hypot>);

NNVM_REGISTER_OP(_backward_hypot)
.set_attr<FCompute>("FCompute<gpu>", BinaryBackwardUseIn<gpu, mshadow_op::hypot_grad_left,
                                          mshadow_op::hypot_grad_right>);

}  // namespace op
}  // namespace mxnet
