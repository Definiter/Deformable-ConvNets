#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2016 by Contributors
 * \file elemwise_unary_op.cu
 * \brief GPU Implementation of unary function.
 */
#include "./elemwise_unary_op.h"
#include "./elemwise_binary_op.h"

namespace mxnet {
namespace op {
// copy
NNVM_REGISTER_OP(_copy)
.set_attr<FCompute>("FCompute<gpu>", IdentityCompute<gpu>);

NNVM_REGISTER_OP(_backward_copy)
.set_attr<FCompute>("FCompute<gpu>", IdentityCompute<gpu>);

NNVM_REGISTER_OP(BlockGrad)
.set_attr<FCompute>("FCompute<gpu>", IdentityCompute<gpu>);

NNVM_REGISTER_OP(make_loss)
.set_attr<FCompute>("FCompute<gpu>", IdentityCompute<gpu>);

// identity output as first input, but attributes are constrainted to be like rhs
NNVM_REGISTER_OP(_identity_with_attr_like_rhs)
.set_attr<FCompute>("FCompute<gpu>", IdentityCompute<gpu>);

NNVM_REGISTER_OP(Cast)
.set_attr<FCompute>("FCompute<gpu>", CastCompute<gpu>);

NNVM_REGISTER_OP(_backward_cast)
.set_attr<FCompute>("FCompute<gpu>", CastCompute<gpu>);

// negative
NNVM_REGISTER_OP(negative)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::negation>);

// abs
NNVM_REGISTER_OP(abs)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::abs>);

NNVM_REGISTER_OP(_backward_abs)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::sign> >);

// sign
NNVM_REGISTER_OP(sign)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::sign>);

NNVM_REGISTER_OP(_backward_sign)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::sign_grad> >);

// round
NNVM_REGISTER_OP(round)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::round>);

// ceil
NNVM_REGISTER_OP(ceil)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::ceil>);

// floor
NNVM_REGISTER_OP(floor)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::floor>);

// rint
NNVM_REGISTER_OP(rint)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::rint>);

// fix
NNVM_REGISTER_OP(fix)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::fix>);

// square
NNVM_REGISTER_OP(square)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::square>);

NNVM_REGISTER_OP(_backward_square)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::square_grad> >);

// sqrt
NNVM_REGISTER_OP(sqrt)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::square_root>);

NNVM_REGISTER_OP(_backward_sqrt)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::square_root_grad> >);

// rsqrt
NNVM_REGISTER_OP(rsqrt)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::reciprocal_square_root>);

NNVM_REGISTER_OP(_backward_rsqrt)
.set_attr<FCompute>("FCompute<gpu>",
  BinaryCompute<gpu, unary_bwd<mshadow_op::reciprocal_square_root_grad> >);

// exp
NNVM_REGISTER_OP(exp)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::exp>);

// log
NNVM_REGISTER_OP(log)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::log>);

// log10
NNVM_REGISTER_OP(log10)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::log10>);

// log2
NNVM_REGISTER_OP(log2)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::log2>);

NNVM_REGISTER_OP(_backward_log)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::log_grad> >);

// log1p
NNVM_REGISTER_OP(log1p)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::log1p>);

NNVM_REGISTER_OP(_backward_log1p)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::log1p_grad> >);

// expm1
NNVM_REGISTER_OP(expm1)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::expm1>);

NNVM_REGISTER_OP(_backward_expm1)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::exp> >);

// sin
NNVM_REGISTER_OP(sin)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::sin>);

NNVM_REGISTER_OP(_backward_sin)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::sin_grad> >);

// cos
NNVM_REGISTER_OP(cos)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::cos>);

NNVM_REGISTER_OP(_backward_cos)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::cos_grad> >);

// tan
NNVM_REGISTER_OP(tan)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::tan>);

NNVM_REGISTER_OP(_backward_tan)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::tan_grad> >);

// arcsin
NNVM_REGISTER_OP(arcsin)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::arcsin>);

NNVM_REGISTER_OP(_backward_arcsin)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::arcsin_grad> >);

// arccos
NNVM_REGISTER_OP(arccos)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::arccos>);

NNVM_REGISTER_OP(_backward_arccos)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::arccos_grad> >);

// arctan
NNVM_REGISTER_OP(arctan)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::arctan>);

NNVM_REGISTER_OP(_backward_arctan)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::arctan_grad> >);

// degrees
NNVM_REGISTER_OP(degrees)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::degrees>);

NNVM_REGISTER_OP(_backward_degrees)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::degrees_grad> >);

// radians
NNVM_REGISTER_OP(radians)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::radians>);

NNVM_REGISTER_OP(_backward_radians)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::radians_grad> >);

// cosh
NNVM_REGISTER_OP(cosh)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::cosh>);

NNVM_REGISTER_OP(_backward_cosh)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::cosh_grad> >);

// sinh
NNVM_REGISTER_OP(sinh)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::sinh>);

NNVM_REGISTER_OP(_backward_sinh)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::sinh_grad> >);

// tanh
NNVM_REGISTER_OP(tanh)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::tanh>);

NNVM_REGISTER_OP(_backward_tanh)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::tanh_grad> >);

// arcsinh
NNVM_REGISTER_OP(arcsinh)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::arcsinh>);

NNVM_REGISTER_OP(_backward_arcsinh)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::arcsinh_grad> >);

// arccosh
NNVM_REGISTER_OP(arccosh)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::arccosh>);

NNVM_REGISTER_OP(_backward_arccosh)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::arccosh_grad> >);

// arctanh
NNVM_REGISTER_OP(arctanh)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::arctanh>);

NNVM_REGISTER_OP(_backward_arctanh)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::arctanh_grad> >);

// gamma
NNVM_REGISTER_OP(gamma)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::gamma>);

NNVM_REGISTER_OP(_backward_gamma)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::gamma_grad> >);

// gammaln
NNVM_REGISTER_OP(gammaln)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::gammaln>);

NNVM_REGISTER_OP(_backward_gammaln)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::gammaln_grad> >);

}  // namespace op
}  // namespace mxnet
