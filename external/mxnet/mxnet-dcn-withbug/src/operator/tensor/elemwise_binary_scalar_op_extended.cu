#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2016 by Contributors
 * \file elemwise_binary_scalar_op.cu
 * \brief GPU Implementation of unary function.
 */
#include "./elemwise_unary_op.h"
#include "./elemwise_binary_op.h"
#include "./elemwise_binary_scalar_op.h"

namespace mxnet {
namespace op {
NNVM_REGISTER_OP(_maximum_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarCompute<gpu, mshadow_op::maximum>);

NNVM_REGISTER_OP(_backward_maximum_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarBackward<gpu, mshadow_op::ge>);

NNVM_REGISTER_OP(_minimum_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarCompute<gpu, mshadow_op::minimum>);

NNVM_REGISTER_OP(_backward_minimum_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarBackward<gpu, mshadow_op::le>);

NNVM_REGISTER_OP(_power_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarCompute<gpu, mshadow_op::power>);

NNVM_REGISTER_OP(_backward_power_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarBackward<gpu, mshadow_op::power_grad>);

NNVM_REGISTER_OP(_rpower_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarCompute<gpu, mshadow_op::rpower>);

NNVM_REGISTER_OP(_backward_rpower_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarBackward<gpu, mshadow_op::rpower_grad>);

NNVM_REGISTER_OP(_hypot_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarCompute<gpu, mshadow_op::hypot>);

NNVM_REGISTER_OP(_backward_hypot_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarBackward<gpu, mshadow_op::hypot_grad_left>);

NNVM_REGISTER_OP(smooth_l1)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarCompute<gpu, mshadow_op::smooth_l1_loss>);

NNVM_REGISTER_OP(_backward_smooth_l1)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarBackward<gpu, mshadow_op::smooth_l1_gradient>);

}  // namespace op
}  // namespace mxnet
