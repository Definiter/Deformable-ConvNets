/*!
 * Copyright (c) 2017 by Contributors
 * \file indexing_op.cu
 * \brief
 * \author Siyi Li, Chi Zhang
*/

#include "./indexing_op.h"
namespace mxnet {
namespace op {
NNVM_REGISTER_OP(Embedding)
.set_attr<FCompute>("FCompute<gpu>", EmbeddingOpForward<gpu>);

NNVM_REGISTER_OP(_backward_Embedding)
.set_attr<FCompute>("FCompute<gpu>", EmbeddingOpBackward<gpu>);

NNVM_REGISTER_OP(take)
.set_attr<FCompute>("FCompute<gpu>", TakeOpForward<gpu>);

NNVM_REGISTER_OP(_backward_take)
.set_attr<FCompute>("FCompute<gpu>", TakeOpBackward<gpu>);

NNVM_REGISTER_OP(batch_take)
.set_attr<FCompute>("FCompute<gpu>", BatchTakeOpForward<gpu>);

NNVM_REGISTER_OP(one_hot)
.set_attr<FCompute>("FCompute<gpu>", OneHotOpForward<gpu>);

}  // namespace op
}  // namespace mxnet

