/*!
 *  Copyright (c) 2016 by Contributors
 * \file elemwise_binary_scalar_op.cu
 * \brief GPU Implementation of unary function.
 */
#include "./elemwise_unary_op.h"
#include "./elemwise_binary_op.h"
#include "./elemwise_binary_broadcast_op.h"

namespace mxnet {
namespace op {
NNVM_REGISTER_OP(broadcast_add)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastCompute<gpu, mshadow::op::plus>);

NNVM_REGISTER_OP(_backward_broadcast_add)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastBackwardUseNone<gpu, mshadow_op::identity,
                                                                mshadow_op::identity>);

NNVM_REGISTER_OP(broadcast_sub)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastCompute<gpu, mshadow::op::minus>);

NNVM_REGISTER_OP(_backward_broadcast_sub)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastBackwardUseNone<gpu, mshadow_op::identity,
                                                                mshadow_op::negation>);

NNVM_REGISTER_OP(broadcast_mul)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastCompute<gpu, mshadow::op::mul>);

NNVM_REGISTER_OP(_backward_broadcast_mul)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastBackwardUseIn<gpu, mshadow_op::right,
                                                                mshadow_op::left>);

NNVM_REGISTER_OP(broadcast_div)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastCompute<gpu, mshadow::op::div>);

NNVM_REGISTER_OP(_backward_broadcast_div)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastBackwardUseIn<gpu, mshadow_op::div_grad,
                                                                mshadow_op::div_rgrad>);

}  // namespace op
}  // namespace mxnet
